#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include "util.h"
#include "opt_2dhisto.h"
#include "ref_2dhisto.h"


__global__ void opt_2dhisto_kernel(uint32_t* d_input, size_t inputSize, unsigned int* d_bins);

void opt_2dhisto(uint32_t* d_input, size_t inputSize, unsigned int* d_bins, hipDeviceProp_t prop)
{
    /* This function should only contain grid setup 
       code and a call to the GPU histogramming kernel. 
       Any memory allocations and transfers must be done 
       outside this function */
    float maxConcurrentBlocks = prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount/(float)prop.maxThreadsPerBlock;
    hipMemset(d_bins, 0, HISTO_HEIGHT*HISTO_WIDTH*sizeof(unsigned int));
    opt_2dhisto_kernel<<<min(ceil(inputSize/(float)prop.maxThreadsPerBlock), maxConcurrentBlocks), prop.maxThreadsPerBlock>>>(d_input, inputSize, d_bins);
    hipDeviceSynchronize();
}

/* Include below the implementation of any other functions you need */

__device__ unsigned int atomicClampedAdd(unsigned int* address, unsigned int value, int limit){
    unsigned int old = *address, assumed;
    do{
      assumed = old;	// READ
      old = atomicCAS(address, assumed, 
            (value + assumed)>limit ? limit : value+assumed);	// MODIFY + WRITE
    } while (assumed != old);
  return old;
}

__global__ void opt_2dhisto_kernel(uint32_t* d_input, size_t inputSize, unsigned int* d_bins){
    __shared__ unsigned int private_bins[HISTO_HEIGHT*HISTO_WIDTH];;
    if(threadIdx.x < HISTO_HEIGHT * HISTO_WIDTH)
        private_bins[threadIdx.x] = 0;
    
    __syncthreads();
    int stride = gridDim.x*blockDim.x;
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    while(index<inputSize){
        atomicAdd(&(private_bins[d_input[index]]), 1);
        index += stride;
    }
    __syncthreads();

    if(threadIdx.x < HISTO_HEIGHT * HISTO_WIDTH)
        atomicClampedAdd(&(d_bins[threadIdx.x]), private_bins[threadIdx.x], UINT8_MAXIMUM);
}